#include "hip/hip_runtime.h"

// OpenCL utility layer include
#include "cmdlineparser.h"
#include "xcl2.hpp"
#include <fcntl.h>
#include <fstream>
#include <iomanip>
#include <iosfwd>
#include <iostream>
#include <unistd.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper for shared that are common to CUDA Samples

size_t max_buffer = 16 * 1024 * 1024;
size_t min_buffer = 4 * 1024;
size_t max_size = 128 * 1024 * 1024; // 128MB

__global__ void vectorCopyGPU(int *a, int *c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        c[idx] = a[idx] + 1;
    }
}

// implicit transfer to GPU device memory from FPGA
void p2p_fpga_to_gpu(cl::Context context,
                     cl::CommandQueue q,
                     cl::Program program,
                     std::vector<int, aligned_allocator<int>> *source_input_A)
{
    int err;
    size_t vector_size_bytes = sizeof(int) * max_buffer;

    // Instantiate FPGA copy kernel
    cl::Kernel krnl;
    cl_mem_ext_ptr_t outExt;
    outExt = {XCL_MEM_EXT_P2P_BUFFER, nullptr, 0};

    OCL_CHECK(err, cl::Buffer input_a(context, CL_MEM_READ_ONLY, vector_size_bytes, nullptr, &err));
    OCL_CHECK(err,
              cl::Buffer p2pBo(context, CL_MEM_WRITE_ONLY | CL_MEM_EXT_PTR_XILINX, vector_size_bytes, &outExt, &err));
    OCL_CHECK(err, krnl = cl::Kernel(program, "bandwidth", &err));

    int *inputPtr = (int *)q.enqueueMapBuffer(input_a, CL_TRUE, CL_MAP_WRITE | CL_MAP_READ, 0, vector_size_bytes,
                                              nullptr, nullptr, &err);

    for (uint32_t i = 0; i < max_buffer; i++)
    {
        inputPtr[i] = i % 1024; // source_input_A[i];
    }
    q.finish();

    // Set the Kernel Arguments
    OCL_CHECK(err, err = krnl.setArg(0, input_a));
    OCL_CHECK(err, err = krnl.setArg(1, p2pBo));
    OCL_CHECK(err, err = krnl.setArg(2, max_buffer));

    OCL_CHECK(err, err = q.enqueueMigrateMemObjects({input_a}, 0 /* 0 means from host*/));
    q.finish();

    // Launch the Kernel
    OCL_CHECK(err, err = q.enqueueTask(krnl));
    q.finish();

    std::cout << "\nMap P2P device buffers to host access pointers\n"
              << std::endl;

    void *p2pPtr1 = nullptr;
    hipMallocManaged(&p2pPtr1, vector_size_bytes);
    p2pPtr1 = q.enqueueMapBuffer(p2pBo,                      // buffer
                                 CL_TRUE,                    // blocking call
                                 CL_MAP_READ | CL_MAP_WRITE, // Indicates we will be writing
                                 0,                          // buffer offset
                                 vector_size_bytes,          // size in bytes
                                 nullptr, nullptr,
                                 &err); // error code
    q.finish();

    std::cout << "Start P2P Read of various buffer sizes from device buffers to GPU\n"
              << std::endl;

    size_t bufsize = min_buffer;
    for (size_t m = 0; m < 1; m++)
    // for (size_t bufsize = min_buffer; bufsize <= vector_size_bytes; bufsize *= 2)
    {
        std::string size_str = xcl::convert_size(bufsize);

        int iter = max_size / bufsize;
        if (xcl::is_emulation())
        {
            iter = 2; // Reducing iteration to run faster in emulation flow.
        }

        // allocate CUDA host and device memory
        int *out = (int *)malloc(bufsize);
        int *d_out;
        hipMalloc(&d_out, bufsize);
        int *out1 = (int *)malloc(bufsize);
        int *d_out1;
        hipMalloc(&d_out1, bufsize);

        // comapre cuda managed input buffer
        int *in = (int *)malloc(bufsize);
        for (int i = 0; i < 10; i++)
        {
            in[i] = i;
        }
        int *d_in;
        hipMalloc(&d_in, bufsize);

        std::chrono::high_resolution_clock::time_point p2pStart = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < 1; i++)
        // for (int i = 0; i < iter; i++)
        {
            // invoke CUDA kernel (copy from one buffer to another)
            int elem = bufsize / sizeof(int);
            dim3 block(256);
            dim3 grid((unsigned int)ceil(elem / (float)block.x));

            // vectorCopyGPU<<<grid, block>>>((int *)p2pPtr1, d_out, elem);
            // hipMemcpy(out, d_out, bufsize, hipMemcpyDeviceToHost);
            // printf("\n===== iter %d ===== \n", i);
            // for (int k = 0; k < 5; k++)
            // {
            //     printf("FPGA kernel input buf %d \n", inputPtr[k]);
            //     printf("FPGA output buffer %d\n", ((int *)p2pPtr1)[k]);
            //     printf("GPU output buffer %d\n", out[k]);
            // }

            // compare CUDA managed input buffer
            printf("\n===== cuda buffer iter %d ===== \n", i);
            hipMemcpy(d_in, in, bufsize, hipMemcpyHostToDevice);
            vectorCopyGPU<<<grid, block>>>(d_in, d_out1, elem);
            hipMemcpy(out1, d_out1, bufsize, hipMemcpyDeviceToHost);
            for (int k = 0; k < 5; k++)
            {
                printf("CUDA input buffer %d\n", in[k]);
                printf("GPU output buffer %d\n", out1[k]);
            }
        }

        std::chrono::high_resolution_clock::time_point p2pEnd = std::chrono::high_resolution_clock::now();
        cl_ulong p2pTime = std::chrono::duration_cast<std::chrono::microseconds>(p2pEnd - p2pStart).count();
        ;
        double dnsduration = (double)p2pTime;
        double dsduration = dnsduration / ((double)1000000);
        double gbpersec = (iter * bufsize / dsduration) / ((double)1024 * 1024 * 1024);
        std::cout << "Buffer = " << size_str << " Iterations = " << iter << " Throughput = " << std::setprecision(2)
                  << std::fixed << gbpersec << "GB/s\n";

        // Free memory
        hipFree(d_out);
        hipFree(d_out1);
        hipFree(d_in);
        free(out);
        free(out1);
        free(in);
    }
}

// implicit transfer to GPU device memory from SSD
int p2p_ssd_to_gpu()
{
    int gpu_n;
    checkCudaErrors(hipGetDeviceCount(&gpu_n));

    // Query device properties
    hipDeviceProp_t prop[64];
    // int gpuid[2]; // we want to find the first two GPU's that can support P2P

    for (int i = 0; i < gpu_n; i++)
    {
        checkCudaErrors(hipGetDeviceProperties(&prop[i], i));
    }
    // Check possibility for peer access
    printf("\nChecking GPU(s) for support of peer to peer memory access...\n");

    int can_access_peer;
    int p2pCapableGPUs[2]; // We take only 1 pair of P2P capable GPUs
    p2pCapableGPUs[0] = p2pCapableGPUs[1] = -1;

    // Show all the combinations of supported P2P GPUs
    for (int i = 0; i < gpu_n; i++)
    {
        for (int j = 0; j < gpu_n; j++)
        {
            if (i == j)
            {
                continue;
            }
            checkCudaErrors(hipDeviceCanAccessPeer(&can_access_peer, i, j));
            printf("> Peer access from %s (GPU%d) -> %s (GPU%d) : %s\n", prop[i].name,
                   i, prop[j].name, j, can_access_peer ? "Yes" : "No");
            if (can_access_peer && p2pCapableGPUs[0] == -1)
            {
                p2pCapableGPUs[0] = i;
                p2pCapableGPUs[1] = j;
            }
        }
    }

    if (p2pCapableGPUs[0] == -1 || p2pCapableGPUs[1] == -1)
    {
        printf(
            "Peer to Peer access is not available amongst GPUs in the system, "
            "waiving test.\n");

        exit(EXIT_WAIVED);
    }

    // Use first pair of p2p capable GPUs detected.
    // gpuid[0] = p2pCapableGPUs[0];
    return 1;
}

int p2p_ssd_to_fpga(int &nvmeFd,
                    cl::Context context,
                    cl::CommandQueue q,
                    cl::Program program,
                    std::vector<int, aligned_allocator<int>> source_input_A)
{

    int err;
    int ret = 0;
    size_t vector_size_bytes = sizeof(int) * max_buffer;

    cl::Kernel krnl;
    // Allocate Buffer in Global Memory
    cl_mem_ext_ptr_t outExt;
    outExt = {XCL_MEM_EXT_P2P_BUFFER, nullptr, 0};

    OCL_CHECK(err, cl::Buffer input_a(context, CL_MEM_READ_ONLY, vector_size_bytes, nullptr, &err));
    OCL_CHECK(err,
              cl::Buffer p2pBo(context, CL_MEM_WRITE_ONLY | CL_MEM_EXT_PTR_XILINX, vector_size_bytes, &outExt, &err));
    OCL_CHECK(err, krnl = cl::Kernel(program, "bandwidth", &err));

    int *inputPtr = (int *)q.enqueueMapBuffer(input_a, CL_TRUE, CL_MAP_WRITE | CL_MAP_READ, 0, vector_size_bytes,
                                              nullptr, nullptr, &err);

    for (uint32_t i = 0; i < max_buffer; i++)
    {
        inputPtr[i] = source_input_A[i];
    }
    q.finish();

    // Set the Kernel Arguments
    OCL_CHECK(err, err = krnl.setArg(0, input_a));
    OCL_CHECK(err, err = krnl.setArg(1, p2pBo));
    OCL_CHECK(err, err = krnl.setArg(2, max_buffer));

    // Copy input data to device global memory
    OCL_CHECK(err, err = q.enqueueMigrateMemObjects({input_a}, 0 /* 0 means from host*/));
    q.finish();

    // Launch the Kernel
    OCL_CHECK(err, err = q.enqueueTask(krnl));
    q.finish();

    std::cout << "\nMap P2P device buffers to host access pointers\n"
              << std::endl;
    void *p2pPtr = q.enqueueMapBuffer(p2pBo,                      // buffer
                                      CL_TRUE,                    // blocking call
                                      CL_MAP_WRITE | CL_MAP_READ, // Indicates we will be writing
                                      0,                          // buffer offset
                                      vector_size_bytes,          // size in bytes
                                      nullptr, nullptr,
                                      &err); // error code
    q.finish();

    std::cout << "Start P2P Write of various buffer sizes from SSD to device buffers\n"
              << std::endl;
    for (size_t bufsize = min_buffer; bufsize <= vector_size_bytes; bufsize *= 2)
    {
        std::string size_str = xcl::convert_size(bufsize);

        int iter = max_size / bufsize;
        if (xcl::is_emulation())
        {
            iter = 2; // Reducing iteration to run faster in emulation flow.
        }
        std::chrono::high_resolution_clock::time_point p2pStart = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < iter; i++)
        {
            ret = pread(nvmeFd, (void *)p2pPtr, bufsize, 0);
            if (ret == -1)
            {
                std::cout << "P2P: read() failed, err: " << ret << ", line: " << __LINE__ << std::endl;
                return EXIT_FAILURE;
            }
        }
        std::chrono::high_resolution_clock::time_point p2pEnd = std::chrono::high_resolution_clock::now();
        cl_ulong p2pTime = std::chrono::duration_cast<std::chrono::microseconds>(p2pEnd - p2pStart).count();
        ;
        double dnsduration = (double)p2pTime;
        double dsduration = dnsduration / ((double)1000000);
        double gbpersec = (iter * bufsize / dsduration) / ((double)1024 * 1024 * 1024);
        std::cout << "Buffer = " << size_str << " Iterations = " << iter << " Throughput = " << std::setprecision(2)
                  << std::fixed << gbpersec << "GB/s\n";
    }
    return 0;
}

void p2p_fpga_to_ssd(int &nvmeFd,
                     cl::Context context,
                     cl::CommandQueue q,
                     cl::Program program,
                     std::vector<int, aligned_allocator<int>> *source_input_A)
{
    int err;
    size_t vector_size_bytes = sizeof(int) * max_buffer;

    // Allocate Buffer in Global Memory
    cl_mem_ext_ptr_t inExt;
    inExt = {XCL_MEM_EXT_P2P_BUFFER, nullptr, 0};

    OCL_CHECK(err, cl::Buffer buffer_input(context, CL_MEM_READ_ONLY | CL_MEM_EXT_PTR_XILINX, vector_size_bytes, &inExt,
                                           &err));

    std::cout << "\nMap P2P device buffers to host access pointers\n"
              << std::endl;
    void *p2pPtr1 = q.enqueueMapBuffer(buffer_input,               // buffer
                                       CL_TRUE,                    // blocking call
                                       CL_MAP_READ | CL_MAP_WRITE, // Indicates we will be writing
                                       0,                          // buffer offset
                                       vector_size_bytes,          // size in bytes
                                       nullptr, nullptr,
                                       &err); // error code
    q.finish();

    std::cout << "Start P2P Read of various buffer sizes from device buffers to SSD\n"
              << std::endl;
    for (size_t bufsize = min_buffer; bufsize <= vector_size_bytes; bufsize *= 2)
    {
        std::string size_str = xcl::convert_size(bufsize);

        int iter = max_size / bufsize;
        if (xcl::is_emulation())
        {
            iter = 2; // Reducing iteration to run faster in emulation flow.
        }
        std::chrono::high_resolution_clock::time_point p2pStart = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < iter; i++)
        {
            if (pwrite(nvmeFd, (void *)p2pPtr1, bufsize, 0) <= 0)
            {
                std::cerr << "ERR: pwrite failed: "
                          << " error: " << strerror(errno) << std::endl;
                exit(EXIT_FAILURE);
            }
        }
        std::chrono::high_resolution_clock::time_point p2pEnd = std::chrono::high_resolution_clock::now();
        cl_ulong p2pTime = std::chrono::duration_cast<std::chrono::microseconds>(p2pEnd - p2pStart).count();
        ;
        double dnsduration = (double)p2pTime;
        double dsduration = dnsduration / ((double)1000000);
        double gbpersec = (iter * bufsize / dsduration) / ((double)1024 * 1024 * 1024);
        std::cout << "Buffer = " << size_str << " Iterations = " << iter << " Throughput = " << std::setprecision(2)
                  << std::fixed << gbpersec << "GB/s\n";
    }
}

int main(int argc, char **argv)
{
    // Command Line Parser
    sda::utils::CmdLineParser parser;

    // Switches
    //**************//"<Full Arg>",  "<Short Arg>", "<Description>", "<Default>"
    parser.addSwitch("--xclbin_file", "-x", "input binary file string", "");
    parser.addSwitch("--file_path", "-p", "file path string", "");
    parser.addSwitch("--input_file", "-f", "input file string", "");
    parser.addSwitch("--device", "-d", "device id", "0");
    parser.parse(argc, argv);

    // Read settings
    auto binaryFile = parser.value("xclbin_file");
    std::string filepath = parser.value("file_path");
    std::string dev_id = parser.value("device");
    std::string filename;

    if (argc < 5)
    {
        parser.printHelp();
        return EXIT_FAILURE;
    }

    if (filepath.empty())
    {
        std::cout << "\nWARNING: As file path is not provided using -p option, going with -f option which is local "
                     "file testing. Please use -p option, if looking for actual p2p operation on NVMe drive.\n";
        filename = parser.value("input_file");
    }
    else
    {
        std::cout << "\nWARNING: Ignoring -f option when -p options is set. -p has high precedence over -f.\n";
        filename = filepath;
    }

    int nvmeFd = -1;
    if (xcl::is_emulation())
    {
        max_buffer = 16 * 1024;
    }

    cl_int err;
    cl::Context context;
    cl::CommandQueue q;
    std::vector<int, aligned_allocator<int>> source_input_A(max_buffer);

    // OPENCL HOST CODE AREA START
    // get_xil_devices() is a utility API which will find the xilinx
    // platforms and will return list of devices connected to Xilinx platform
    auto devices = xcl::get_xil_devices();
    // read_binary_file() is a utility API which will load the binaryFile
    // and will return the pointer to file buffer.
    auto fileBuf = xcl::read_binary_file(binaryFile);
    cl::Program::Binaries bins{{fileBuf.data(), fileBuf.size()}};
    cl::Program program;

    auto pos = dev_id.find(":");
    cl::Device device;
    if (pos == std::string::npos)
    {
        uint32_t device_index = stoi(dev_id);
        if (device_index >= devices.size())
        {
            std::cout << "The device_index provided using -d flag is outside the range of "
                         "available devices\n";
            return EXIT_FAILURE;
        }
        device = devices[device_index];
    }
    else
    {
        if (xcl::is_emulation())
        {
            std::cout << "Device bdf is not supported for the emulation flow\n";
            return EXIT_FAILURE;
        }
        device = xcl::find_device_bdf(devices, dev_id);
    }

    if (xcl::is_hw_emulation())
    {
        auto device_name = device.getInfo<CL_DEVICE_NAME>();
        if (device_name.find("2018") != std::string::npos)
        {
            std::cout << "[INFO]: The example is not supported for " << device_name
                      << " this platform for hw_emu. Please try other flows." << '\n';
            return EXIT_SUCCESS;
        }
    }

    // Creating Context and Command Queue for selected Device
    OCL_CHECK(err, context = cl::Context(device, nullptr, nullptr, nullptr, &err));
    OCL_CHECK(err, q = cl::CommandQueue(context, device, CL_QUEUE_PROFILING_ENABLE, &err));
    std::cout << "Trying to program device[" << dev_id << "]: " << device.getInfo<CL_DEVICE_NAME>() << std::endl;
    program = cl::Program(context, {device}, bins, nullptr, &err);
    if (err != CL_SUCCESS)
    {
        std::cout << "Failed to program device[" << dev_id << "] with xclbin file!\n";
        exit(EXIT_FAILURE);
    }
    else
        std::cout << "Device[" << dev_id << "]: program successful!\n";

    // P2P transfer from SSD to FPGA
    std::cout << "############################################################\n";
    std::cout << "                  Reading data from SSD                       \n";
    std::cout << "############################################################\n";
    // Get access to the NVMe SSD.
    nvmeFd = open(filename.c_str(), O_RDWR | O_DIRECT);
    if (nvmeFd < 0)
    {
        std::cerr << "ERROR: open " << filename << "failed: " << std::endl;
        return EXIT_FAILURE;
    }
    std::cout << "INFO: Successfully opened NVME SSD " << filename << std::endl;
    int ret = 0;
    ret = p2p_ssd_to_fpga(nvmeFd, context, q, program, source_input_A);
    (void)close(nvmeFd);
    if (ret != 0)
        return EXIT_FAILURE;

    // P2P transfer from FPGA to SSD
    std::cout << "############################################################\n";
    std::cout << "                  Writing data to SSD                       \n";
    std::cout << "############################################################\n";

    nvmeFd = open(filename.c_str(), O_RDWR | O_DIRECT);
    if (nvmeFd < 0)
    {
        std::cerr << "ERROR: open " << filename << "failed: " << std::endl;
        return EXIT_FAILURE;
    }
    std::cout << "INFO: Successfully opened NVME SSD " << filename << std::endl;
    p2p_fpga_to_ssd(nvmeFd, context, q, program, &source_input_A);
    (void)close(nvmeFd);

    // P2P transfer from FPGA to GPU
    std::cout << "############################################################\n";
    std::cout << "       GPU CUDA kernel accessing FPGA memory                \n";
    std::cout << "############################################################\n";
    p2p_fpga_to_gpu(context, q, program, &source_input_A);

    std::cout << "TEST PASSED" << std::endl;
    return EXIT_SUCCESS;
}
